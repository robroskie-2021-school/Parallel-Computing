
#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>
#include <time.h>


#define CHK(x){                  \
    hipError_t e = x;           \
    if(e != hipSuccess){        \
        printf("\nCUDA Error" + e);  \
        hipDeviceReset();       \
        exit(1);                 \
}}                               \



__global__ void addKernel(double *b, double n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        b[i] = (double)i / n;
}

int main()
{   
    double start, end;
    int n = 10000000;
    double size = n * sizeof(double);
   
    //Serial code
    double* a;
    a = (double*)malloc(size);
    
    //Initialize array and compute time taken
    start = clock();
    for (int i = 0; i < n; i++)
        a[i] = (double)i / n;
    end = clock();

    //Print test results and time taken
    for (int j = 0; j < 5; j++)
        printf("a[%d]: %.7f\n", j, a[j]);
    printf("...\n");
    for (int j = n-5; j < n; j++)
        printf("a[%d]: %.7f\n", j, a[j]);

    printf("Serial code took %.2f ms to initialize array\n\n",(end - start) / CLOCKS_PER_SEC * 1000);

    //Repeat parallelized version of procedure 
    double* b;
    double* b_d;

    b = (double*)malloc(n * sizeof(double));
    
    CHK(hipMalloc(&b_d, size));

    CHK(hipMemcpy(b_d, b, size, hipMemcpyHostToDevice));

    start = clock();
    //n = 10000 * 1000
    addKernel<<<10000,1000>>>(b_d, n);
    hipDeviceSynchronize();
    end = clock();

    CHK(hipMemcpy(b, b_d, size, hipMemcpyDeviceToHost));
 
    for (int j = 0; j < 5; j++)
        printf("b[%d]: %.7f\n", j, b[j]);
    printf("...\n");
    for (int j = n - 5; j < n; j++)
        printf("b[%d]: %.7f\n", j, b[j]);

    printf("Parallel code took %.2f ms to initialize array", (end - start) / CLOCKS_PER_SEC * 1000);

    return 0;
}
